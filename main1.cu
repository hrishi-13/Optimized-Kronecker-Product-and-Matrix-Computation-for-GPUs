/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-1
 * Description: Computation of a matrix C = Kronecker_prod(A, B.T)
 *              where A and B are matrices of dimension (m, n) and
 *              the output is of the dimension (m * n, m * n). 
 * Note: All lines marked in --> should be replaced with code. 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; // The handle for printing the output

__global__ void per_row_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    long int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < m * m) {
        // printf("%ld ", id);
        long int i = id / m;
        long int j = id % m;
        // printf("%ld, %ld\n", i, j);

        for(long int p = 0; p < n; p++) {
            for(long int q = 0; q < n; q++) {
                C[(n*i + q)* m*n + m*p + j] = A[i*n + p] * B[j*n + q];
                // C[n*i + q][m*p + j] = A[i][p] * B[j][q] 
            }
        }
    }
}

__global__ void per_column_AB_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    long int id = (blockIdx.x * 32 * 32) + (threadIdx.y * 32) + threadIdx.x;
    if(id < n*n) { 
        // printf("%ld ", id);
        long int i = id / n;   
        long int j = id % n;
        // printf("%ld, %ld\n", i, j);

        for(long int p = 0; p < m; p++) {
          for(long int q = 0; q < m; q++) {
            C[(n*p + j)*m*n + m*i + q] = A[p*n + i] * B[q*n + j];
            // C[n*p + j][m*i + q] = A[p][i] * B[q][j]
          }
        }
    }
}

__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){
    long int id = (gridDim.y * blockDim.x * blockDim.y * blockIdx.x) + (blockDim.x * blockDim.y * blockIdx.y) + (threadIdx.x * blockDim.y) + threadIdx.y;
    if(id < (m*m*n*n)) {
        // printf("%ld ", id);
        long int i = id / (m*n);
        long int j = id % (m*n);
        // printf("%ld %ld\n", i, j);

        C[i*m*n + j] = A[(i/n)*n + j/m] * B[(j%m)*n + i%n];
        // C[i][j] = A[i/n][j/m] * B[j%m][i%n]
    }
}

// Prints any 1D array in the form of a matrix

void printMatrix(long int *arr, long int rows, long int cols, char* filename){
    outfile.open(filename);
    for(long int i = 0; i < rows; i++){
        for(long int j = 0; j < cols; j++){
            outfile<<arr[i * cols + j]<<" ";
        }
        outfile<<"\n";
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/

double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc, char **argv){
    // Variable declarations
    long int m,n;	
    cin >> m >> n;	

    // Host_arrays 
    long int *h_a, *h_b, *h_c;

    // Device arrays 
    long int *d_a, *d_b, *d_c;
	
    /* Allocating space for the host_arrays */
    h_a = (long int *) malloc(m * n * sizeof(long int));
    h_b = (long int *) malloc(m * n * sizeof(long int));	
    h_c = (long int *) malloc(m * m * n * n * sizeof(long int));	

    /* Allocating memory for the device arrays */
    // Allocating memory for A on device 
    hipMalloc(&d_a, m*n*sizeof(long int));

    // Allocating memory for B on device 
    hipMalloc(&d_b, m*n*sizeof(long int));

    // Allocating memory for C on device 
    hipMalloc(&d_c, m*n*n*m*sizeof(long int));

    // Reading the input matrix A 
    for(long int i = 0; i < m * n; i++) {
        cin >> h_a[i];
    }

    // Reading the input matrix B 
    for(long int i = 0; i < m * n; i++) {
        cin >> h_b[i];
    }

    // Transfer the input host arrays to the device 
    // Copying A from Host to Device
    // Copying B from Host to Device 
    hipMemcpy(d_a, h_a, m*n*sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, m*n*sizeof(long int), hipMemcpyHostToDevice);

    long int gridDimx, gridDimy;
    
    /**
     * Kernel 1 - per_row_AB_kernel
     * To be launched with 1D grid, 1D block
     * Each thread should process a complete row of A, B
     **/

    // Setting the launch configuration 
    gridDimx =  ceil((float)(m*m)/1024);
    dim3 grid1(gridDimx, 1, 1); // 1D Grid
    dim3 block1(1024, 1, 1); // 1D Block

    double starttime = rtclock();  

    // Launching the per_row kernel 
    per_row_AB_kernel<<<grid1, block1>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize();                                                           

    double endtime = rtclock(); 
	printtime("GPU Kernel-1 time: ", starttime, endtime);  

    // Copying C from Device to Host 
    hipMemcpy(h_c, d_c, m*n*m*n*sizeof(long int), hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel1.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 2 - per_column_AB_kernel
     * To be launched with 1D grid, 2D block
     * Each thread should process a complete column of  A, B
     **/
    
    // Setting the launch configuration 
    gridDimx =  ceil((float)(n*n)/1024);
    dim3 grid2(gridDimx, 1, 1); // 1D Grid
    dim3 block2(32, 32, 1); // 2D Block

    starttime = rtclock(); 

    // Launching the per_column kernel 
    per_column_AB_kernel<<<grid2, block2>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize(); 

    endtime = rtclock(); 
  	printtime("GPU Kernel-2 time: ", starttime, endtime);  

    // Copying C from Device to Host
    hipMemcpy(h_c, d_c, m*n*m*n*sizeof(long int), hipMemcpyDeviceToHost);

    printMatrix(h_c, m * n, m * n,"kernel2.txt");
    hipMemset(d_c, 0, m * n * m * n * sizeof(long int));

    /**
     * Kernel 3 - per_element_kernel
     * To be launched with 2D grid, 2D block
     * Each thread should process one element of the output 
     **/

    // Setting the launch configuration 
    gridDimx = ceil(float(n * n) / 16);
    gridDimy = ceil(float(m * m) / 64);
    dim3 grid3(gridDimx,gridDimy,1); // 2D Grid
    dim3 block3(64,16,1); // 2D Block

    starttime = rtclock();  

    // Launching the per_element kernel 
    per_element_kernel<<<grid3, block3>>>(d_a, d_b, d_c, m, n);
    hipDeviceSynchronize();                                                              

    endtime = rtclock();  
	printtime("GPU Kernel-3 time: ", starttime, endtime);  

    // Copying C from Device to Host
    hipMemcpy(h_c, d_c, m*n*m*n*sizeof(long int), hipMemcpyDeviceToHost);
    printMatrix(h_c, m * n, m * n, "kernel3.txt");

    return 0;
}
